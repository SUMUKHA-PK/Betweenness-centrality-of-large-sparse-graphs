#include "hip/hip_runtime.h"
#include "graphs.cuh"

__global__
void bfs(int * done, Vertex * vs, Edge * es, int no_nodes)
{        
    long long int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(vid < no_nodes)
    {
        if(vs[id].inQ == true && vs[id].visited == false)
        {
            
            vs[id].visited = true;

            vs[id].inQ = false;

            __syncthreads();
            
            Item * temp = es[id].item;
            
            while( temp->item != NULL)
            {
                int n_id = temp.id;

                if(vs[n_id].visited == false)
                {
                    *done = false;

                    if(vs[n_id].distance == vs[id].distance + 1)
                    {
                        Item * t = vs[n_id].item;
                        while(t->item != NULL)
                        {
                            t = t->item;
                        }
                        t->id = id;
                        Item nItem = new Item; 
                        *nItem = {.id : -1, .item : NULL};
                        t->item = nItem;
                    }
                    else if(vs[n_id].distance > vs[id].distance + 1)
                    {
                        vs[n_id].item->id = id;
                        Item nItem = new Item; 
                        *nItem = {.id : -1, .item : NULL};    
                        vs[n_id].item->item = nItem;
                    }
                }
            }
        }
    }
}

namespace graphs
{
    void graph::call_bfs(int ** G)
    {
        Vertex * h_vs = new Vertex[5];
        Vertex * d_vs;
        hipMalloc((void **)&d_vs, 5 * sizeof(Vertex));

        Edge * h_es = new Edge[5]; 
        Edge * d_es;
        hipMalloc((void **)&d_es, 5 * sizeof(Edge));

        for(int i=0; i < 5; i++){

            Item * temp = new Item;
            *temp = {.id : -1, .item : NULL};
            h_vs[i].item = temp;
            h_es[i].item = temp;

            h_vs[i].distance = MAX;
            h_vs[i].visited = false;
            h_vs[i].inQ = false;

            for(int j=0; j < 5; j++){
                if(G[i][j] == 1){
                    temp = h_es[i].item;

                    while(temp->item != NULL){
                        temp = temp->item;
                    }

                    Item * tempE = new Item;
                    *tempE = {.id : j, .item : NULL};
                    temp->item = tempE;
                }
            }

            h_vs[0].distance = 0;
            h_vs[0].inQ = 1;
            
            int done;
            hipMemcpy(d_vs, h_vs, 5 * sizeof(Vertex), hipMemcpyHostToDevice);
            hipMemcpy(d_es, h_es, 5 * sizeof(Edge), hipMemcpyHostToDevice);
            
            bool h_done = true, * d_done;
            hipMalloc((void **)&d_done, sizeof(bool));

            bfs<<<5, 5>>>(d_done, d_vs, d_es, 5);
        }
    }
}