#include "hip/hip_runtime.h"
#include "graphs.cuh"

__global__
void stage1(int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist, Edge * d_edges){        
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    for(id=0;i<d_q_curlen;i++)
    {
        for(int i=0;i<edges[id].no_neigh;i++)
        {
            if(atomicCAS(d_))
        }
    }
}

namespace graphs{

    void calculateBC(Edge * h_edges, int no_nodes){

        int * d_q_curlen, * d_q_nexlen, * d_depth, * d_S_len, * d_ends_len;

        int * d_q_cur, * d_q_next, * d_sigma, * d_delta, * d_S, * d_ends, * d_dist;
        
        Edge * d_edges;

        hipMalloc((void **)&d_q_curlen, sizeof(int));
        hipMalloc((void **)&d_q_nexlen, sizeof(int));
        hipMalloc((void **)&d_depth, sizeof(int));
        hipMalloc((void **)&d_S_len, sizeof(int));
        hipMalloc((void **)&d_ends_len, sizeof(int));

        hipMalloc((void **)&d_q_cur, no_nodes*sizeof(int));
        hipMalloc((void **)&d_q_next, no_nodes*sizeof(int));
        hipMalloc((void **)&d_sigma, no_nodes*sizeof(int));
        hipMalloc((void **)&d_delta, no_nodes*sizeof(int));
        hipMalloc((void **)&d_S, no_nodes*sizeof(int));
        hipMalloc((void **)&d_ends, no_nodes*sizeof(int));
        hipMalloc((void **)&d_dist, no_nodes*sizeof(int));

        hipMalloc((void **)&d_edges, no_nodes*sizeof(Edge));

        int One = 1;
        int Zero = 0;

        // Initialize
        hipMemcpy(d_q_curlen, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_q_nexlen, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S_len, &One, sizeof(int, hipMemcpyHostToDevice));
        hipMemcpy(d_ends_len, &One, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_q_cur, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_sigma, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, &Zero, sizeof(int), hipMemcpyHostToDevice);
           
        hipMemcpy(d_edges, h_edges, no_nodes*sizeof(Edge), hipMemcpyHostToDevice);

    }
}