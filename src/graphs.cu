#include "hip/hip_runtime.h"
#include "graphs.cuh"
using namespace graphs;

#define size 8

__global__
void stage1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, int * d_no_nodes, Edge * d_edges){        
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    if(id<*d_q_curlen)
    {   
        int current = d_q_cur[id];
        for(int i=0;i<d_edges[current].no_neigh;i++)
        {
            if(atomicCAS(&d_dist[d_edges[current].neighbours[i]],INT_MAX,d_dist[current]+1)==INT_MAX)
            {
                int temp = atomicAdd(d_q_nexlen,1);
                d_q_next[temp]=d_edges[current].neighbours[i];
            }
            if(d_dist[d_edges[current].neighbours[i]]==(d_dist[current]+1))
                atomicAdd(&d_sigma[d_edges[current].neighbours[i]],d_sigma[current]);
        }
        __syncthreads();
    }
}

__global__
void stage1_1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, int * d_no_nodes, Edge * d_edges)
{
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    if(id<*d_q_nexlen)
    {
        d_q_cur[id]=d_q_next[id];
        d_S[id+*d_S_len]=d_q_next[id];
        __syncthreads();
    }
}

__global__ 
void single(int * d_depth, int * d_dist, int * d_S, int * d_S_len){
    *d_depth=d_dist[d_S[*d_S_len-1]]-1;
}

__global__ 
void singleThread(int * d_ends, int * d_ends_len, int * d_q_nexlen, int * d_q_curlen, int * d_S_len){
    d_ends[*d_ends_len]=d_ends[*d_ends_len-1]+*d_q_nexlen;
    *d_ends_len = *d_ends_len + 1;
    *d_q_curlen=*d_q_nexlen;
    *d_S_len+=*d_q_nexlen;
    *d_q_nexlen=0;
}

__global__
void stage2_2(int * d_delta, int *  d_dist, int *  d_sigma, int * d_S, Edge * d_edges, const int offset, const int itr){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx <= itr){
        int tid = idx + offset;
        int w = d_S[tid];
        float dsw = 0;
        int sw = d_sigma[w];
    
        for(int i = 0; i < d_edges[w].no_neigh; i++){
            int v = d_edges[w].neighbours[i];
            if(d_dist[v] == d_dist[w] + 1){
                dsw += ((float) sw * (1 + d_delta[v])) / d_sigma[v];
            }
        }
        d_delta[w] = (int)dsw;

        __syncthreads();
    }
}


namespace graphs{

    void calculateBC(Edge * h_edges, int no_nodes){

        int * d_q_curlen, * d_q_nexlen, * d_depth, * d_S_len, * d_ends_len, * d_no_nodes, h_q_nexlen;

        int * d_q_cur, * d_q_next, * d_sigma, * d_delta, *h_delta, * d_S, * d_ends, * d_dist, * h_ends, h_depth;

        int * h_dis = new int[no_nodes];
        h_ends = new int[no_nodes];

        for(int cc=0;cc<no_nodes;cc++)
        {
            h_ends[cc] = 0;
            h_dis[cc] = INT_MAX;
        }

        h_dis[0] = 0;
        
        Edge * d_edges;

        bool * d_status;

        hipMalloc((void **)&d_q_curlen, sizeof(int));
        hipMalloc((void **)&d_q_nexlen, sizeof(int));
        hipMalloc((void **)&d_depth, sizeof(int));
        hipMalloc((void **)&d_S_len, sizeof(int));
        hipMalloc((void **)&d_ends_len, sizeof(int));
        hipMalloc((void **)&d_no_nodes, sizeof(int));
        hipMalloc((void **)&d_status, sizeof(bool));

        hipMalloc((void **)&d_q_cur, no_nodes*sizeof(int));
        hipMalloc((void **)&d_q_next, no_nodes*sizeof(int));
        hipMalloc((void **)&d_sigma, no_nodes*sizeof(int));
        hipMalloc((void **)&d_delta, no_nodes*sizeof(int));
        hipMalloc((void **)&d_S, no_nodes*sizeof(int));
        hipMalloc((void **)&d_ends, no_nodes*sizeof(int));
        hipMalloc((void **)&d_dist, no_nodes*sizeof(int));

        hipMalloc((void **)&d_edges, no_nodes*sizeof(Edge));

        hipMemset(d_delta, 0, no_nodes*sizeof(int));

        int One = 1;
        int Zero = 0;
        int Two = 2;
        int initEnd[2] = {0, 1};

        // Initialize
        hipMemcpy(d_q_curlen, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_q_nexlen, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S_len, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends_len, &Two, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends, initEnd, 2*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_no_nodes, &no_nodes, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_q_cur, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_sigma, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, h_dis, no_nodes*sizeof(int), hipMemcpyHostToDevice);
           
        hipMemcpy(d_edges, h_edges, no_nodes*sizeof(Edge), hipMemcpyHostToDevice);

        while(1){
            stage1<<<10,10>>>(d_status,d_q_curlen,d_q_nexlen,d_S_len,d_ends_len,d_q_cur,d_q_next,d_sigma,d_delta,d_S,d_ends,d_dist,d_depth,d_no_nodes,d_edges);
            hipMemcpy(&h_q_nexlen, d_q_nexlen, sizeof(int), hipMemcpyDeviceToHost);
 
            if(h_q_nexlen==0){
                single<<<1, 1>>>(d_depth, d_dist, d_S, d_S_len);
                break;
            }
            stage1_1<<<10,10>>>(d_status,d_q_curlen,d_q_nexlen,d_S_len,d_ends_len,d_q_cur,d_q_next,d_sigma,d_delta,d_S,d_ends,d_dist,d_depth,d_no_nodes,d_edges);
            singleThread<<<1, 1>>>(d_ends, d_ends_len, d_q_nexlen, d_q_curlen, d_S_len);
        }
        
        hipMemcpy(&h_depth,d_depth,sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(h_ends,d_ends,no_nodes * sizeof(int),hipMemcpyDeviceToHost);
        
        int counter = h_depth;

        int offset;

        while(counter >= 0){

            offset = h_ends[counter];
            int itr = h_ends[counter + 1] - 1 - offset;

            // int blocks = ceil((float)itr/size);

            stage2_2<<<1, size>>>(d_delta, d_dist, d_sigma, d_S, d_edges, (const int)offset, (const int)itr);

            counter --;
        }

        h_delta = new int[no_nodes];
        
        hipMemcpy(h_delta, d_delta, no_nodes * sizeof(int),hipMemcpyDeviceToHost);
        
        hipFree(d_q_curlen);
        hipFree(d_q_nexlen);
        hipFree(d_depth);
        hipFree(d_S_len);
        hipFree(d_ends_len);

        hipFree(d_q_cur);
        hipFree(d_q_next);
        hipFree(d_sigma);
        hipFree(d_delta);
        hipFree(d_S);
        hipFree(d_ends);
        hipFree(d_dist);
    }
}