#include "hip/hip_runtime.h"
#include "graphs.cuh"

namespace graph
{

    graph::__global__ void bfs(int * done, Vertex * vs, Edge * es, int no_nodes)
    {        
        long long int id = threadIdx.x + blockIdx.x * blockDim.x;

        if(vid < no_nodes)
        {
            if(vs[id].inQ == true && vs[id].visited == false)
            {
                
                vs[id].visited = true;

                vs[id].inQ = false;

                __syncthreads();
                
                Item * temp = es[id].item;
                
                while( temp->item != NULL)
                {
                    int n_id = temp.id;

                    if(vs[n_id].visited == false)
                    {
                        *done = false;

                        if(vs[n_id].distance == vs[id].distance + 1)
                        {
                            Item * t = vs[n_id].item;
                            while(t->item != NULL)
                            {
                                t = t->item;
                            }
                            t->id = id;
                            Item nItem = new Item; 
                            *nItem = {.id : -1, .item : NULL};
                            t->item = nItem;
                        }
                        else if(vs[n_id].distance > vs[id].distance + 1)
                        {
                            vs[n_id].item->id = id;
                            Item nItem = new Item; 
                            *nItem = {.id : -1, .item : NULL};    
                            vs[n_id].item->item = nItem;
                        }
                    }
                }
            }
        }
    }
}