#include "hip/hip_runtime.h"
#include "graphs.cuh"
using namespace graphs;

__global__
void stage1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, Edge * d_edges){        
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    
    for(int i=0;i<d_edges[id].no_neigh;i++)
    {
        if(atomicCAS(&d_dist[i],INT_MAX,d_dist[id]+1)==INT_MAX)
        {
            int temp = atomicAdd(d_q_nexlen,1);
            d_q_next[temp]=i;
        }
        if(d_dist[i]==(d_dist[id]+1))
            atomicAdd(&d_sigma[i],d_sigma[id]);
    }

    __syncthreads();

    if(*d_q_nexlen==0)
    {
        *d_depth=d_dist[d_S[*d_S_len-1]]-1;
        *status = false;
    }
    else
    {
        if(id>=0&&id<*d_q_nexlen)
        {
            d_q_cur[id]=d_q_next[id];
            d_S[id+*d_S_len]=d_q_next[id];
        }
        __syncthreads();
        
        d_ends[*d_ends_len]=d_ends[*d_ends_len-1]+*d_q_nexlen;
        *d_ends_len++;
        *d_q_curlen=*d_q_nexlen;
        *d_S_len+=*d_q_nexlen;
        *d_q_nexlen=0;

        __syncthreads();
    }
}

__global__
void stage1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, Edge * d_edges){        
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
}

namespace graphs{

    void calculateBC(Edge * h_edges, int no_nodes){

        int * d_q_curlen, * d_q_nexlen, * d_depth, * d_S_len, * d_ends_len;

        int * d_q_cur, * d_q_next, * d_sigma, * d_delta, * d_S, * d_ends, * d_dist, h_depth;
        
        Edge * d_edges;

        bool h_status, * d_status;

        hipMalloc((void **)&d_q_curlen, sizeof(int));
        hipMalloc((void **)&d_q_nexlen, sizeof(int));
        hipMalloc((void **)&d_depth, sizeof(int));
        hipMalloc((void **)&d_S_len, sizeof(int));
        hipMalloc((void **)&d_ends_len, sizeof(int));
        hipMalloc((void **)&d_status, sizeof(bool));

        hipMalloc((void **)&d_q_cur, no_nodes*sizeof(int));
        hipMalloc((void **)&d_q_next, no_nodes*sizeof(int));
        hipMalloc((void **)&d_sigma, no_nodes*sizeof(int));
        hipMalloc((void **)&d_delta, no_nodes*sizeof(int));
        hipMalloc((void **)&d_S, no_nodes*sizeof(int));
        hipMalloc((void **)&d_ends, no_nodes*sizeof(int));
        hipMalloc((void **)&d_dist, no_nodes*sizeof(int));

        hipMalloc((void **)&d_edges, no_nodes*sizeof(Edge));

        int One = 1;
        int Zero = 0;

        // Initialize
        hipMemcpy(d_q_curlen, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_q_nexlen, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S_len, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends_len, &One, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_q_cur, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_sigma, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, &Zero, sizeof(int), hipMemcpyHostToDevice);
           
        hipMemcpy(d_edges, h_edges, no_nodes*sizeof(Edge), hipMemcpyHostToDevice);

        while(1){
            h_status = true;
            hipMemcpy(d_status, &h_status, sizeof(bool), hipMemcpyHostToDevice);
            stage1<<<10,10>>>(d_status,d_q_curlen,d_q_nexlen,d_S_len,d_ends_len,d_q_cur,d_q_next,d_sigma,d_delta,d_S,d_ends,d_dist,d_depth,d_edges);
            hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);

            if(h_status == false)
                break;
        }
        
        hipMemcpy(&h_depth,d_depth,sizeof(int),hipMemcpyDeviceToHost);

        hipFree(d_q_curlen);
        hipFree(d_q_nexlen);
        hipFree(d_depth);
        hipFree(d_S_len);
        hipFree(d_ends_len);

        hipFree(d_q_cur);
        hipFree(d_q_next);
        hipFree(d_sigma);
        hipFree(d_delta);
        hipFree(d_S);
        hipFree(d_ends);
        hipFree(d_dist);

        int counter = h_depth;


        while(counter--){
            stage2<<<10,10>>>();
        }
    }
}