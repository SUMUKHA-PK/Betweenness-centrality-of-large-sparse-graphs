#include "hip/hip_runtime.h"
#include "graphs.cuh"
using namespace graphs;

#define size 8

__global__
void stage1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, int * d_no_nodes, Edge * d_edges){        
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    if(id<*d_no_nodes)
    {
        for(int i=0;i<d_edges[id].no_neigh;i++)
        {
            if(atomicCAS(&d_dist[d_edges[id].neighbours[i]],INT_MAX,d_dist[id]+1)==INT_MAX)
            {
                printf("%d netlen,id %d \n",*d_q_nexlen,id);
                int temp = atomicAdd(d_q_nexlen,1);
                printf("%d netlen,id %d\n",*d_q_nexlen,id);
                d_q_next[temp]=d_edges[id].neighbours[i];
            }
            if(d_dist[d_edges[id].neighbours[i]]==(d_dist[id]+1))
                atomicAdd(&d_sigma[d_edges[id].neighbours[i]],d_sigma[id]);
        }

        __syncthreads();
    }
}

__global__
void stage1_1(bool * status, int * d_q_curlen, int * d_q_nexlen, int * d_S_len, int * d_ends_len, int * d_q_cur, int * d_q_next, int * d_sigma, int * d_delta, int * d_S, int * d_ends, int * d_dist,int* d_depth, int * d_no_nodes, Edge * d_edges)
{
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    if(id<*d_no_nodes)
    {
        printf("%d netlen,id %d\n",*d_q_nexlen,id);
        if(*d_q_nexlen==0)
        {
            *d_depth=d_dist[d_S[*d_S_len-1]]-1;
            *status = false;
        }
        else
        {
            if(id>=0&&id<*d_q_nexlen)
            {
                d_q_cur[id]=d_q_next[id];
                d_S[id+*d_S_len]=d_q_next[id];
            }
            __syncthreads();
            
            d_ends[*d_ends_len]=d_ends[*d_ends_len-1]+*d_q_nexlen;
            *d_ends_len++;
            *d_q_curlen=*d_q_nexlen;
            *d_S_len+=*d_q_nexlen;
            *d_q_nexlen=0;

            __syncthreads();
        }

        if(id==0)
        {
            for(int x = 0;x<5;x++)
                printf("%d ",d_sigma[x]);
            printf("sigma\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_ends[x]);
            printf("ends\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_S[x]);
            printf("S\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_dist[x]);
            printf("dist\n");
            for(int x=0;x<*d_q_curlen;x++)
                printf("%d ",d_q_cur[x]);
            printf("cur\n");
            for(int x=0;x<*d_q_nexlen;x++)
                printf("%d ",d_q_next[x]);
            printf("nex\n");
        }
        if(id==1)
        {
            for(int x = 0;x<5;x++)
                printf("%d ",d_sigma[x]);
            printf("sigma\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_ends[x]);
            printf("ends\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_S[x]);
            printf("S\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_dist[x]);
            printf("dist\n");
            for(int x=0;x<*d_q_curlen;x++)
                printf("%d ",d_q_cur[x]);
            printf("cur\n");
            for(int x=0;x<*d_q_nexlen;x++)
                printf("%d ",d_q_next[x]);
            printf("nex\n");
        }
        if(id==2)
        {
            for(int x = 0;x<5;x++)
                printf("%d ",d_sigma[x]);
            printf("sigma\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_ends[x]);
            printf("ends\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_S[x]);
            printf("S\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_dist[x]);
            printf("dist\n");
            for(int x=0;x<*d_q_curlen;x++)
                printf("%d ",d_q_cur[x]);
            printf("cur\n");
            for(int x=0;x<*d_q_nexlen;x++)
                printf("%d ",d_q_next[x]);
            printf("nex\n");
        }
        if(id==3)
        {
            for(int x = 0;x<5;x++)
                printf("%d ",d_sigma[x]);
            printf("sigma\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_ends[x]);
            printf("ends\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_S[x]);
            printf("S\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_dist[x]);
            printf("dist\n");
            for(int x=0;x<*d_q_curlen;x++)
                printf("%d ",d_q_cur[x]);
            printf("cur\n");
            for(int x=0;x<*d_q_nexlen;x++)
                printf("%d ",d_q_next[x]);
            printf("nex\n");
        }
        if(id==4)
        {
            for(int x = 0;x<5;x++)
                printf("%d ",d_sigma[x]);
            printf("sigma\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_ends[x]);
            printf("ends\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_S[x]);
            printf("S\n");
            for(int x = 0;x<5;x++)
                printf("%d ",d_dist[x]);
            printf("dist\n");
            for(int x=0;x<*d_q_curlen;x++)
                printf("%d ",d_q_cur[x]);
            printf("cur\n");
            for(int x=0;x<*d_q_nexlen;x++)
                printf("%d ",d_q_next[x]);
            printf("nex\n");
        }
    }
}
__global__
void stage2(int * d_delta, int *  d_dist, int *  d_sigma, int * d_S, Edge * d_edges, int offset,int itr){        
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < itr){
        int tid = idx + offset;
        int w = d_S[tid];
        float dsw = 0;
        int sw = d_sigma[w];

        for(int i = 0; i < d_edges[w].no_neigh; i++){
            int v = d_edges[w].neighbours[i];
            if(d_dist[] == d_dist[w] + 1){
                dsw += (float) sw * (1 + d_delta[v]) / d_sigma[v];
            }
        }
        d_delta[w] = (int)dsw;
        
        __syncthreads();
    }
}

namespace graphs{

    void calculateBC(Edge * h_edges, int no_nodes){

        int * d_q_curlen, * d_q_nexlen, * d_depth, * d_S_len, * d_ends_len, * d_no_nodes;

        int * d_q_cur, * d_q_next, * d_sigma, * d_delta, * d_S, * d_ends, * d_dist, * h_ends, h_depth;

        int * h_dis = new int[no_nodes];

        for(int cc=0;cc<no_nodes;cc++)
        {
            h_dis[cc]=INT_MAX;
        }

        h_dis[0] = 0;

        for(int cc=0;cc<no_nodes;cc++)
        {
            printf("%d ",h_dis[cc]);
        }
        
        Edge * d_edges;

        bool h_status, * d_status;

        hipMalloc((void **)&d_q_curlen, sizeof(int));
        hipMalloc((void **)&d_q_nexlen, sizeof(int));
        hipMalloc((void **)&d_depth, sizeof(int));
        hipMalloc((void **)&d_S_len, sizeof(int));
        hipMalloc((void **)&d_ends_len, sizeof(int));
        hipMalloc((void **)&d_no_nodes, sizeof(int));
        hipMalloc((void **)&d_status, sizeof(bool));

        hipMalloc((void **)&d_q_cur, no_nodes*sizeof(int));
        hipMalloc((void **)&d_q_next, no_nodes*sizeof(int));
        hipMalloc((void **)&d_sigma, no_nodes*sizeof(int));
        hipMalloc((void **)&d_delta, no_nodes*sizeof(int));
        hipMalloc((void **)&d_S, no_nodes*sizeof(int));
        hipMalloc((void **)&d_ends, no_nodes*sizeof(int));
        hipMalloc((void **)&d_dist, no_nodes*sizeof(int));

        hipMalloc((void **)&d_edges, no_nodes*sizeof(Edge));

        hipMemset(d_delta, 0, no_nodes*sizeof(int));

        int One = 1;
        int Zero = 0;

        // Initialize
        hipMemcpy(d_q_curlen, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_q_nexlen, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S_len, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends_len, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_no_nodes, &no_nodes, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_q_cur, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_sigma, &One, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_S, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ends, &Zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, h_dis, no_nodes*sizeof(int), hipMemcpyHostToDevice);
           
        hipMemcpy(d_edges, h_edges, no_nodes*sizeof(Edge), hipMemcpyHostToDevice);

        while(1){
            cout << "Hi " << endl;
            h_status = true;
            hipMemcpy(d_status, &h_status, sizeof(bool), hipMemcpyHostToDevice);
            stage1<<<10,10>>>(d_status,d_q_curlen,d_q_nexlen,d_S_len,d_ends_len,d_q_cur,d_q_next,d_sigma,d_delta,d_S,d_ends,d_dist,d_depth,d_no_nodes,d_edges);
            stage1_1<<<10,10>>>(d_status,d_q_curlen,d_q_nexlen,d_S_len,d_ends_len,d_q_cur,d_q_next,d_sigma,d_delta,d_S,d_ends,d_dist,d_depth,d_no_nodes,d_edges);
            hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
            printf("rwlb\ns %d \n",h_status);
            if(h_status == false)
                break;
        }
        
        hipMemcpy(&h_depth,d_depth,sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(h_ends,d_ends,no_nodes * sizeof(int),hipMemcpyDeviceToHost);

        int counter = h_depth;

        while(counter >= 0){
            int offset = ends[depth];
            int itr = ends[depth + 1] - 1 - offset;

            int blocks = ceil((float)itr/size);
            stage2<<<blocks,size>>>(d_delta, d_dist, d_sigma, d_S, d_edges, offset, itr);
            counter --;
        }

        hipFree(d_q_curlen);
        hipFree(d_q_nexlen);
        hipFree(d_depth);
        hipFree(d_S_len);
        hipFree(d_ends_len);

        hipFree(d_q_cur);
        hipFree(d_q_next);
        hipFree(d_sigma);
        hipFree(d_delta);
        hipFree(d_S);
        hipFree(d_ends);
        hipFree(d_dist);
    }
}