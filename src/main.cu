//Sumukha Pk 16CO145
//Prajval M  16CO234

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<vector>
#include<string>
#include"graphs.cuh"

using namespace std;
using namespace graph;

int main(int argc,char ** argv)
{
    FILE *fp;
    fp = fopen(argv[argc-1], "r");
    char c = fgetc(fp); 
    int flag = -1;
    long long int i,j,a=0,nnz=2,b=0,count=0;
    while (c != EOF) 
    { 
        if(c=='\n')
        {
            if(flag==-1)
            {
                a=count;
                count=0;
                flag++;
            }
            else if(flag==0)
            {
                b=count;
                count=0;
                flag++;
            }
        }
        else if(c==' ') ++count;
        c = fgetc(fp); 
    } 
    fclose(fp);
    fp = fopen(argv[argc-1], "r");
    c = fgetc(fp);
    long long int *n1 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n2 = (long long int *)malloc(b*sizeof(long long int));
    long long int *n3 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n4 = (long long int *)malloc(nnz*sizeof(long long int));
    count=0;
    string x = "";
    while (c != EOF) 
    { 
        if(c!=' ')
        {
            x+=c;
        } 
        else if(c==' '||c=='\n')
        {
            if(count<a)
            {
                n1[count]=stoi(x);
                x="";
                count++;
            }   
            else if(count>=a&&count<(a+b))
            {
                n2[count-(a)]=stoi(x);
                x="";
                count++;
            }
            else if(count>=(a+b)&&count<(2*a+b))
            {
                n3[count-(a+b)]=stoi(x);
                x="";
                count++;
            }
            else
            {
                n4[count-(2*a+b)]=stoi(x);
                x="";
                count++;
            }
        }
       c = fgetc(fp); 
    } 
    n4[1]=stoi(x);
    fclose(fp);

//-------------------------------------------File input done------------------------------

    long long int dim2=n4[0],dim1=n4[1];
    Edge * edges = new Edge[dim1];   //Number of edges is size of n1
    Vertex * vertices = new Vertex[dim2];

    for(i=0;i<dim1;i++)
        edges[i]->item=NULL;
    
    long long cc =0,cd=0,ignore=0;
    for(i=1;i<=dim1;i++)
    {
        vertices[i-1].id=i-1;
        long long x = n2[i]-n2[i-1];
        for(j=cc;j<cc+x;j++)
        {
            if((i-1)!=n3[j])
            {
                edges[i-1].from=(i-1);
                edges[cd].to=n3[j];
                cd++;
                Item * temp = edges[i-1].item;
                while(temp->item!=NULL)
                    temp=temp->item;
                Item *t = new Item;
                temp->item=t;
                t->item=NULL;
            }
            else
            {   
                ignore++;
            }
        }
        cc+=x;
    }
    a-=ignore;
    // for(i=0;i<a;i++)
    // {
    //     cout<<edges[i].from<<" "<<edges[i].to<<endl;
    // }
    
    for(i=a;i<2*a;i++)
    {
        edges[i].from=edges[i-a].to;
        edges[i].to=edges[i-a].from;
    }
    free(n1);
    free(n2);
    free(n3);
    free(n4);
//---------------------------------------------Graph is generated--------------------------

    
    return 0;
}