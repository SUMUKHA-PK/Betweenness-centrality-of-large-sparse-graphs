#include "hip/hip_runtime.h"
//Sumukha Pk 16CO145
//Prajval M  16CO234

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<vector>
#include<string>
#include <ctime>
#include"graphs.cuh"
#include"graphs.cu"

using namespace std;
using namespace graphs;

void printDevProp(hipDeviceProp_t devProp){
    
    // printf("Compute Capability:            %d.%d\n",  devProp.major, devProp.minor);
    printf("Device Name:                   %s\n",  devProp.name);
    // printf("Total global memory:           %zu\n",  devProp.totalGlobalMem);
    // printf("Total shared memory per block: %zu\n",  devProp.sharedMemPerBlock);
    // printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    // printf("Warp size:                     %d threads\n",  devProp.warpSize);
    // printf("Maximum memory pitch:          %zu\n",  devProp.memPitch);
    // printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    // for (int i = 0; i < 3; ++i)
    // 	printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    // for (int i = 0; i < 3; ++i)
    // 	printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    // printf("Total constant memory:         %zu\n",  devProp.totalConstMem);
    // printf("Texture alignment:             %zu\n",  devProp.textureAlignment);
    // printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    // printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    // printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int main(int argc,char ** argv)
{
    FILE *fp;
    fp = fopen(argv[argc-1], "r");
    char c = fgetc(fp); 
    int flag = -1;
    long long int i,j,a=0,nnz=2,b=0,count=0;
    while (c != EOF) 
    { 
        if(c=='\n')
        {
            if(flag==-1)
            {
                a=count;
                count=0;
                flag++;
            }
            else if(flag==0)
            {
                b=count;
                count=0;
                flag++;
            }
        }
        else if(c==' ') ++count;
        c = fgetc(fp); 
    } 
    fclose(fp);
    fp = fopen(argv[argc-1], "r");
    c = fgetc(fp);
    long long int *n1 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n2 = (long long int *)malloc(b*sizeof(long long int));
    long long int *n3 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n4 = (long long int *)malloc(nnz*sizeof(long long int));
    count=0;
    string x = "";
    while (c != EOF) 
    { 
        if(c!=' ')
        {
            x+=c;
        } 
        else if(c==' '||c=='\n')
        {
            if(count<a)
            {
                n1[count]=stoi(x);
                x="";
                count++;
            }   
            else if(count>=a&&count<(a+b))
            {
                n2[count-(a)]=stoi(x);
                x="";
                count++;
            }
            else if(count>=(a+b)&&count<(2*a+b))
            {
                n3[count-(a+b)]=stoi(x);
                x="";
                count++;
            }
            else
            {
                n4[count-(2*a+b)]=stoi(x);
                x="";
                count++;
            }
        }
       c = fgetc(fp); 
    } 
    n4[1]=stoi(x);
    fclose(fp);

//-------------------------------------------File input done------------------------------

    long long int dim1=n4[1];
    Edge * edges = new Edge[dim1];   //Number of edges is size of n1
    
    long long cc =0,ignore=0;
    for(i=1;i<=dim1;i++)
    {
        edges[i-1].no_neigh=0;
        long long x = n2[i]-n2[i-1];
        for(j=cc;j<cc+x;j++)
        {
            if((i-1)!=n3[j])
            {
                edges[i-1].neighbours[edges[i-1].no_neigh] = n3[j];
                edges[i-1].no_neigh+=1;
            }
            else
            {   
                ignore++;
            }
        }
        cc+=x;
    }
    a-=ignore;

    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    printDevProp(devProp);

    cout << "Started Computing ...." << endl << endl;

    clock_t begin = clock();

    calculateBC(edges, dim1);    

    clock_t end = clock();
    
    double elapsed_secs = double(end - begin) / (CLOCKS_PER_SEC * 1000);

    cout << "Elapsed Time : " << elapsed_secs << endl;

    cout << endl << "Completed Computing ...." << endl;

    free(n1);
    free(n2);
    free(n3);
    free(n4);
//---------------------------------------------Graph is generated--------------------------

    
    return 0;
}