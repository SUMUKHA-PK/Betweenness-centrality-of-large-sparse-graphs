#include "hip/hip_runtime.h"
//Sumukha Pk 16CO145
//Prajval M  16CO234

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<vector>
#include<string>
#include"graphs.cuh"

using namespace std;
using namespace graphs;

__global__ void kernel(int *d,int *Q,int *Q2,int *R,int *sigma)
{
    int n = 5;
    int s=0;

    int idx = threadIdx.x;
    //Initialize d and sigma
    for(int k=idx; k<n; k+=blockDim.x) 
    {
        if(k == s)
        {
            d[k] = 0;
            sigma[k] = 1;
        }
        else
        {
            d[k] = INT_MAX;
            sigma[k] = 0;
        }
    }

    __shared__ int Q_len;
    __shared__ int Q2_len;

    if(idx == 0)
    {
        Q[0] = s;
        Q_len = 1;
        Q2_len = 0;
    }
    __syncthreads();

    while(1)
    {
        for(int k=idx; k<Q_len; k+=blockDim.x)
        {
            int v = Q[k];
            for(int w=R[v]; w<R[v+1]; w++)
            {
                // Use atomicCAS to prevent duplicates
                if(atomicCAS(&d[w],INT_MAX,d[v]+1) == INT_MAX)
                {
                    int t = atomicAdd(&Q2_len,1);
                    Q2[t] = w;
                }
                if(d[w] == (d[v]+1))
                {
                    atomicAdd(&sigma[w],sigma[v]);
                }
            }
        }
        __syncthreads();

        if(Q2_len == 0)
        {
            //The next vertex frontier is empty, so we're done searching
            break;
        }
        else
        {
            for(int k=idx; k<Q2_len; k+=blockDim.x)
            {
                Q[k] = Q2[k];
            }

            __syncthreads();

            if(idx == 0)
            {
                Q_len = Q2_len;
                Q2_len = 0;
            }
            __syncthreads();
        }
    }
}

int main(int argc,char ** argv)
{
    FILE *fp;
    fp = fopen(argv[argc-1], "r");
    char c = fgetc(fp); 
    int flag = -1;
    long long int i,j,a=0,nnz=2,b=0,count=0;
    while (c != EOF) 
    { 
        if(c=='\n')
        {
            if(flag==-1)
            {
                a=count;
                count=0;
                flag++;
            }
            else if(flag==0)
            {
                b=count;
                count=0;
                flag++;
            }
        }
        else if(c==' ') ++count;
        c = fgetc(fp); 
    } 
    fclose(fp);
    fp = fopen(argv[argc-1], "r");
    c = fgetc(fp);
    long long int *n1 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n2 = (long long int *)malloc(b*sizeof(long long int));
    long long int *n3 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n4 = (long long int *)malloc(nnz*sizeof(long long int));
    count=0;
    string x = "";
    while (c != EOF) 
    { 
        if(c!=' ')
        {
            x+=c;
        } 
        else if(c==' '||c=='\n')
        {
            if(count<a)
            {
                n1[count]=stoi(x);
                x="";
                count++;
            }   
            else if(count>=a&&count<(a+b))
            {
                n2[count-(a)]=stoi(x);
                x="";
                count++;
            }
            else if(count>=(a+b)&&count<(2*a+b))
            {
                n3[count-(a+b)]=stoi(x);
                x="";
                count++;
            }
            else
            {
                n4[count-(2*a+b)]=stoi(x);
                x="";
                count++;
            }
        }
       c = fgetc(fp); 
    } 
    n4[1]=stoi(x);
    fclose(fp);

//-------------------------------------------File input done------------------------------

    long long int dim2=n4[0],dim1=n4[1];
    Edge * edges = new Edge[dim1];   //Number of edges is size of n1
    Vertex * vertices = new Vertex[dim2];

    for(i=0;i<dim1;i++)
        edges[i]->item=NULL;
    
    long long cc =0,cd=0,ignore=0;
    for(i=1;i<=dim1;i++)
    {
        vertices[i-1].id=i-1;
        long long x = n2[i]-n2[i-1];
        for(j=cc;j<cc+x;j++)
        {
            if((i-1)!=n3[j])
            {
                edges[i-1].from=(i-1);
                edges[cd].to=n3[j];
                cd++;
                Item * temp = edges[i-1].item;
                while(temp->item!=NULL)
                    temp=temp->item;
                Item *t = new Item;
                temp->item=t;
                t->item=NULL;
            }
            else
            {   
                ignore++;
            }
        }
        cc+=x;
    }
    a-=ignore;
    // for(i=0;i<a;i++)
    // {
    //     cout<<edges[i].from<<" "<<edges[i].to<<endl;
    // }
    
    for(i=a;i<2*a;i++)
    {
        edges[i].from=edges[i-a].to;
        edges[i].to=edges[i-a].from;
    }
    free(n1);
    free(n2);
    free(n3);
    free(n4);
//---------------------------------------------Graph is generated--------------------------

    
    return 0;
}