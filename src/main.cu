#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/extrema.h>

#include "hip/hip_runtime.h"
#include ""

#define THREADS 64

using namespace std;
using namespace thrust;

typedef struct vertex {
    int parents[15];
    int no_parents;
    int distance;
    bool done;
} Vertex;

typedef struct qu {
    int id;
    int distance;
} Queue;

typedef struct edge{
    int tos[10];
    int no_to;
} Edge;

struct compare_key_value{
    __host__ __device__
    bool operator()(Queue l, Queue r) {
        return l.distance < r.distance;
    }
};

struct compare_key_value qSelector;

int ExtractMin(device_vector<Queue> &queue){

    device_vector<Queue>::iterator t = min_element(queue.begin(), queue.end(), qSelector);

    Queue x = *t;
    
    queue.erase(t);

    return x.id;
}

__global__
void relax(Vertex * nodes, int * edges, int id, int dist, int no_edges){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx < no_edges){
        if(nodes[idx].done == false){
            if(nodes[idx].distance > dist + 1){
                nodes[idx].distance = dist;
                nodes[idx].parents[nodes[idx].no_parents] = id;
                nodes[idx].no_parents = 1;
            }
            else if(nodes[idx].distance == dist + 1){
                nodes[idx].parents[nodes[idx].no_parents - 1] = id;
                nodes[idx].no_parents += 1;
            }
        }
    }

}

int dijkstra(int source, int target, int no_nodes, vector<Edge> es){

    host_vector<Vertex> h_nodes(no_nodes);
    host_vector<Queue> h_queue(no_nodes);
    host_vector<Edge> h_edges(es);
    host_vector<Queue> temp(no_nodes);

    for(int i = 0; i < no_nodes; i++){
        h_queue[i].id = i;
        h_queue[i].distance = INT_MAX;
        h_nodes[i].distance = INT_MAX;
        h_nodes[i].no_parents = 0;
        h_nodes[i].done = true;
    }

    h_queue[source].distance = 0;
    h_nodes[source].distance = 0;

    device_vector<Vertex> d_nodes(h_nodes);
    device_vector<Queue> d_queue(h_queue);
    device_vector<Edge> d_edges(h_edges);

    Vertex * d_nodes_ptr = raw_pointer_cast(d_nodes.data());
    Edge * d_edges_ptr = raw_pointer_cast(d_edges.data());
    
    for(int i = 0; i < no_nodes - 1; i++){
        int id = ExtractMin(d_queue);
        int no_edges = h_edges[id].no_to;
        int * edges;

        hipMalloc((void **)edges, no_edges*sizeof(int));
        cudaMemCpy(edges, )
    }

    return 0;
}

int main(){

    int B[5][5] = {
        {0, 1, 0, 1, 0},
        {1, 0, 1, 0, 0},
        {0, 1, 0, 1, 1},
        {1, 0, 1, 0, 0},
        {0, 0, 1, 0, 0},
    };

    vector<Edge> es; 

    for(int i=0; i < 5; i++){
        Edge temp;
        temp.no_to = 0;

        es.push_back(temp);

        for(int j=0; j < 5; j++){
            if(B[i][j] == 1){
                es[i].tos[es[i].no_to] = j;
                es[i].no_to += 1;
            }
        }
    }

    dijkstra(0, 4, 5, es);
}