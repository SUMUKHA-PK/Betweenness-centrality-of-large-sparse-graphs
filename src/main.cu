//Sumukha Pk 16CO145
//Prajval M  16CO234

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<vector>
#include<string>
using namespace std;

int main()
{
    FILE *fp;
    fp = fopen("file.txt", "r");
    char c = fgetc(fp); 
    int flag = -1;
    long long int i,j,a=0,nnz=2,b=0,count=0;
    while (c != EOF) 
    { 
        if(c=='\n')
        {
            if(flag==-1)
            {
                a=count;
                count=0;
                flag++;
            }
            else if(flag==0)
            {
                b=count;
                count=0;
                flag++;
            }
        }
        else if(c==' ') ++count;
        c = fgetc(fp); 
    } 
    fclose(fp);
    fp = fopen("file.txt", "r");
    c = fgetc(fp);
    long long int *n1 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n2 = (long long int *)malloc(b*sizeof(long long int));
    long long int *n3 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n4 = (long long int *)malloc(nnz*sizeof(long long int));
    count=0;
    string x = "";
    while (c != EOF) 
    { 
        if(c!=' ')
        {
            x+=c;
        } 
        else if(c==' '||c=='\n')
        {
            if(count<a)
            {
                n1[count]=stoi(x);
                x="";
                count++;
            }   
            else if(count>=a&&count<(a+b))
            {
                n2[count-(a)]=stoi(x);
                x="";
                count++;
            }
            else if(count>=(a+b)&&count<(2*a+b))
            {
                n3[count-(a+b)]=stoi(x);
                x="";
                count++;
            }
            else
            {
                n4[count-(2*a+b)]=stoi(x);
                x="";
                count++;
            }
        }
       c = fgetc(fp); 
    } 
    n4[1]=stoi(x);
    fclose(fp);

//-------------------------------------------File input done------------------------------

    long long int dim2=n4[0],dim1=n4[1];
    long long int *graph = (long long int *)malloc(dim1*dim2*sizeof(long long int));
    for(i=0;i<dim1;i++)
    {
        for(j=0;j<dim2;j++)
            *(graph+i*dim2+j)=0;
    }
    int cc=0;
    for(i=1;i<=dim1;i++)
    {
        long long int x = n2[i]-n2[i-1];
        for(j=cc;j<cc+x;j++)
        {
            *(graph+(i-1)*dim2+n3[j])=n1[j];
        }
        cc+=x;
    }
    for(i=0;i<dim1;i++)
    {
        for(j=0;j<dim2;j++)
            printf("%lld ",*(graph+i*dim2+j));
        printf("\n");
    }

//---------------------------------------------Graph is generated--------------------------

    
    return 0;
}