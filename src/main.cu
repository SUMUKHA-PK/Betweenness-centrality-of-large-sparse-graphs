//Sumukha Pk 16CO145
//Prajval M  16CO234

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<vector>
#include<string>
#include"graphs.cuh"

using namespace std;
using namespace graphs;

int main(int argc,char ** argv)
{
    FILE *fp;
    fp = fopen(argv[argc-1], "r");
    char c = fgetc(fp); 
    int flag = -1;
    long long int i,j,a=0,nnz=2,b=0,count=0;
    while (c != EOF) 
    { 
        if(c=='\n')
        {
            if(flag==-1)
            {
                a=count;
                count=0;
                flag++;
            }
            else if(flag==0)
            {
                b=count;
                count=0;
                flag++;
            }
        }
        else if(c==' ') ++count;
        c = fgetc(fp); 
    } 
    fclose(fp);
    fp = fopen(argv[argc-1], "r");
    c = fgetc(fp);
    long long int *n1 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n2 = (long long int *)malloc(b*sizeof(long long int));
    long long int *n3 = (long long int *)malloc(a*sizeof(long long int));
    long long int *n4 = (long long int *)malloc(nnz*sizeof(long long int));
    count=0;
    string x = "";
    while (c != EOF) 
    { 
        if(c!=' ')
        {
            x+=c;
        } 
        else if(c==' '||c=='\n')
        {
            if(count<a)
            {
                n1[count]=stoi(x);
                x="";
                count++;
            }   
            else if(count>=a&&count<(a+b))
            {
                n2[count-(a)]=stoi(x);
                x="";
                count++;
            }
            else if(count>=(a+b)&&count<(2*a+b))
            {
                n3[count-(a+b)]=stoi(x);
                x="";
                count++;
            }
            else
            {
                n4[count-(2*a+b)]=stoi(x);
                x="";
                count++;
            }
        }
       c = fgetc(fp); 
    } 
    n4[1]=stoi(x);
    fclose(fp);

//-------------------------------------------File input done------------------------------

    long long int dim1=n4[1];
    Edge * edges = new Edge[dim1];   //Number of edges is size of n1
    
    long long cc =0,ignore=0;
    for(i=1;i<=dim1;i++)
    {
        edges[i-1].no_neigh=0;
        long long x = n2[i]-n2[i-1];
        for(j=cc;j<cc+x;j++)
        {
            if((i-1)!=n3[j])
            {
                edges[i-1].neighbours[edges[i-1].no_neigh] = n3[j];
                edges[i-1].no_neigh+=1;
            }
            else
            {   
                ignore++;
            }
        }
        cc+=x;
    }
    a-=ignore;

    for(int i = 0; i < dim1; i++){
        cout << edges[i].no_neigh << endl;

        for(int j=0; j<edges[i].no_neigh; j++)
            cout << edges[i].neighbours[j] << "\t";

        cout << endl << endl;
    }

    free(n1);
    free(n2);
    free(n3);
    free(n4);
//---------------------------------------------Graph is generated--------------------------

    
    return 0;
}