#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/extrema.h>

#include "hip/hip_runtime.h"
#include ""

#define THREADS 64

using namespace std;
using namespace thrust;

typedef struct vertex {
    int parents[15];
    int no_parents;
    int distance;
    bool done;
} Vertex;

typedef struct qu {
    int id;
    int distance;
} Queue;

typedef struct edge{
    int tos[10];
    int no_to;
} Edge;

struct compare_key_value{
    __host__ __device__
    bool operator()(Queue l, Queue r) {
        return l.distance < r.distance;
    }
};

struct compare_key_value qSelector;

int ExtractMin(device_vector<Queue> &queue){

    device_vector<Queue>::iterator t = min_element(queue.begin(), queue.end(), qSelector);

    Queue x = *t;
    
    cout << x.distance << endl;

    queue.erase(t);

    return x.id;
}

__global__
void relax(Vertex * nodes, int * edges, int id, int no_edges){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    nodes[id].done = true;

    __syncthreads();

    if(idx < no_edges){
        if(nodes[edges[idx]].done == false){
            // printf("%d %d %d %d\n", nodes[edges[idx]].distance, nodes[id].distance + 1, edges[idx], id);
            if(nodes[edges[idx]].distance == nodes[id].distance + 1){
                nodes[edges[idx]].parents[nodes[edges[idx]].no_parents - 1] = id;
                nodes[edges[idx]].no_parents += 1;
            }
            if(nodes[edges[idx]].distance > nodes[id].distance + 1){
                nodes[edges[idx]].distance = nodes[id].distance + 1;
                nodes[edges[idx]].parents[nodes[edges[idx]].no_parents] = id;
                nodes[edges[idx]].no_parents = 1;
            }
        }
    }

}

int dijkstra(int source, int target, int no_nodes, vector<Edge> es){

    host_vector<Vertex> h_nodes(no_nodes);
    host_vector<Queue> h_queue(no_nodes);
    host_vector<Edge> h_edges(es);

    for(int i = 0; i < no_nodes; i++){
        h_queue[i].id = i;
        h_queue[i].distance = INT_MAX;
        h_nodes[i].distance = INT_MAX;
        h_nodes[i].no_parents = 0;
        h_nodes[i].done = false;
    }

    h_queue[source].distance = 0;
    h_nodes[source].distance = 0;

    device_vector<Vertex> d_nodes(h_nodes);
    device_vector<Queue> d_queue(h_queue);

    Vertex * d_nodes_ptr = raw_pointer_cast(d_nodes.data());
    
    for(int i = 0; i < no_nodes - 1; i++){
        int id = ExtractMin(d_queue);
        cout << "ID  :" << id << endl; 
        int no_edges = h_edges[id].no_to;
        int * edges;

        hipMalloc((void **)&edges, no_edges*sizeof(int));
        hipMemcpy(edges, h_edges[id].tos, no_edges*sizeof(int), hipMemcpyHostToDevice);

        for(int i=0; i<no_edges; i++){
            cout << h_edges[id].tos[i] << "\t";
        }
        
        int blocks = ceil((float)no_edges/THREADS);
        relax<<<blocks, THREADS>>>(d_nodes_ptr, edges, id, no_edges);
        cout << endl << endl;
    }
    cout << endl << endl;

    for(int i=0; i < no_nodes; i++){
        Vertex temp = d_nodes[i];
        cout << i << " " << temp.distance << " " << temp.no_parents << " " << endl;
    }

    return 0;
}

int main(){

    int B[5][5] = {
        {0, 1, 0, 1, 0},
        {1, 0, 1, 0, 0},
        {0, 1, 0, 1, 1},
        {1, 0, 1, 0, 0},
        {0, 0, 1, 0, 0},
    };

    vector<Edge> es; 

    for(int i=0; i < 5; i++){
        Edge temp;
        temp.no_to = 0;

        es.push_back(temp);

        for(int j=0; j < 5; j++){
            if(B[i][j] == 1){
                es[i].tos[es[i].no_to] = j;
                es[i].no_to += 1;
            }
        }
    }

    dijkstra(0, 4, 5, es);
}