#include<hip/hip_runtime.h>
#include<iostream>
using namespace std;

int n = 5;
int s=0;

__global__ void kernel(int *d,int *Q,int *Q2,int *R,int *sigma)
{
    int n = 5;
    int s=0;

    int idx = threadIdx.x;
    //Initialize d and sigma
    for(int k=idx; k<n; k+=blockDim.x) 
    {
        if(k == s)
        {
            d[k] = 0;
            sigma[k] = 1;
        }
        else
        {
            d[k] = INT_MAX;
            sigma[k] = 0;
        }
    }

    if(idx==4)
    {
        for(int i=0;i<5;i++)
            printf("%d ",d[i]);
        printf("D done\n");
        for(int i=0;i<5;i++)
            printf("%d ",Q[i]);
        printf("Q done\n");
        for(int i=0;i<5;i++)
            printf("%d ",Q2[i]);
        printf("Q2 done\n");
        for(int i=0;i<5;i++)
            printf("%d ",R[i]);
        printf("R done\n");
        for(int i=0;i<5;i++)
            printf("%d ",sigma[i]);
        printf("Sigma done\n");
    }
    

    __shared__ int Q_len;
    __shared__ int Q2_len;

    if(idx == 0)
    {
        Q[0] = s;
        Q_len = 1;
        Q2_len = 0;
    }
    __syncthreads();

    while(1)
    {
        for(int k=idx; k<Q_len; k+=blockDim.x)
        {
            int v = Q[k];
            for(int w=R[v]; w<R[v+1]; w++)
            {
                // Use atomicCAS to prevent duplicates
                if(atomicCAS(&d[w],INT_MAX,d[v]+1) == INT_MAX)
                {
                    int t = atomicAdd(&Q2_len,1);
                    Q2[t] = w;
                }
                if(d[w] == (d[v]+1))
                {
                    atomicAdd(&sigma[w],sigma[v]);
                }
            }
        }
        __syncthreads();

        if(Q2_len == 0)
        {
            //The next vertex frontier is empty, so we're done searching
            break;
        }
        else
        {
            for(int k=idx; k<Q2_len; k+=blockDim.x)
            {
                Q[k] = Q2[k];
            }

            __syncthreads();

            if(idx == 0)
            {
                Q_len = Q2_len;
                Q2_len = 0;
            }
            __syncthreads();
        }
    }
}

int main()
{
    int d[5]={0,0,0,0,0};
    int Q[5]={0,1,2,3,4};
    int Q2[5]={1,2,3,4,5};
    int R[5]={3,4,1,2,0};
    int sigma[5]={1,2,3,4,5};

    int *d_d,*d_Q,*d_Q2,*d_R,*d_sigma;

    hipMalloc((void **)&d_d, 5 * sizeof(int));
    hipMalloc((void **)&d_Q, 5 * sizeof(int));
    hipMalloc((void **)&d_Q2, 5 * sizeof(int));
    hipMalloc((void **)&d_R, 5 * sizeof(int));
    hipMalloc((void **)&d_sigma, 5 * sizeof(int));

    hipMemcpy(d_d, d, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Q, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Q2, Q2, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sigma, sigma, 5 * sizeof(int), hipMemcpyHostToDevice);
    
    kernel<<<5,5>>>(d_d,d_Q,d_Q2,d_R,d_sigma);


    hipMemcpy(d, d_d, 5*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Q, d_Q, 5*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Q2, d_Q2, 5*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(R, d_R, 5*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(sigma, d_sigma, 5*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<5;i++)
        cout<<d[i]<<" ";
    cout<<endl;
    for(int i=0;i<5;i++)
        cout<<Q[i]<<" ";
    cout<<endl;
    for(int i=0;i<5;i++)
        cout<<Q2[i]<<" ";
    cout<<endl;
    for(int i=0;i<5;i++)
        cout<<R[i]<<" ";
    cout<<endl;
    for(int i=0;i<5;i++)
        cout<<sigma[i]<<" ";
    cout<<endl;
    return 0;
}