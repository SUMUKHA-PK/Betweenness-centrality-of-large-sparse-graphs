
#include <hip/hip_runtime.h>
#include <iostream>
#define MAX 1000000

using namespace std;

typedef struct vertex
{
    int parents[10];
    int no_parents;
    bool visited;
    bool inQ;
    long long distance;
} Vertex;

typedef struct edge
{
    int tos[10];
    int no_to;
} Edge;

void call_bfs(int ** G);

__device__ void lock(int *mutex) {
	while (atomicCAS(mutex, 0, 1) != 0);
}

__device__ void unlock(int *mutex) {
	atomicExch(mutex, 0);
}

__global__
void bfs(bool * done, Vertex * vs, Edge * es, int no_nodes, int * mutex);

int main()
{ 
    int ** G = new int * [5];
    int B[5][5] = {
        {0, 1, 0, 1, 0},
        {1, 0, 1, 0, 0},
        {0, 1, 0, 1, 1},
        {1, 0, 1, 0, 0},
        {0, 0, 1, 0, 0},
    };
    
    for(int i = 0; i < 5; i++)
    {
        G[i] = new int[5];
        for(int j=0; j < 5; j++)
            G[i][j] = B[i][j];
    }

    call_bfs(G);

    return 0;
}

void call_bfs(int ** G)
{
    Vertex * h_vs = new Vertex[5];
    Vertex * d_vs;
    hipMalloc((void **)&d_vs, 5 * sizeof(Vertex));

    Edge * h_es = new Edge[5]; 
    Edge * d_es;
    hipMalloc((void **)&d_es, 5 * sizeof(Edge));

    for(int i=0; i < 5; i++)
    {
        h_vs[i].no_parents = 0;
        h_es[i].no_to = 0;

        h_vs[i].distance = MAX;
        h_vs[i].visited = false;
        h_vs[i].inQ = false;

        for(int j=0; j < 5; j++)
        {
            if(G[i][j] == 1)
            {
                h_es[i].tos[h_es[i].no_to] = j;
                h_es[i].no_to += 1;
            }
        }
    }

    h_vs[0].distance = 0;
    h_vs[0].inQ = 1;
    
    hipMemcpy(d_vs, h_vs, 5 * sizeof(Vertex), hipMemcpyHostToDevice);
    hipMemcpy(d_es, h_es, 5 * sizeof(Edge), hipMemcpyHostToDevice);
    
    bool h_done, *d_done;
    hipMalloc((void **)&d_done, sizeof(bool));

    int * mutex;
    hipMalloc((void **)&mutex, sizeof(int));

    while(true){
        h_done = true;
        hipMemcpy(d_done, &h_done, sizeof(bool), hipMemcpyHostToDevice);
        bfs<<<5, 5>>>(d_done, d_vs, d_es, 5, mutex);
        hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
        if(h_done == true)
            break;
    }
    
    hipMemcpy(h_vs, d_vs, 5 * sizeof(Vertex), hipMemcpyDeviceToHost);

    for(int i=0; i < 5; i++){
        Vertex node = h_vs[i];
        cout << "Node :" << i << endl;
        cout << "Distance: " << node.distance << endl << "No Of Parents: " << node.no_parents << endl << "Parents :";
        for(int j = 0; j < node.no_parents; j++){
            cout << node.parents[j] << "\t";
        }
        cout << endl << endl;
    }
}

__global__
void bfs(bool * done, Vertex * vs, Edge * es, int no_nodes, int * mutex)
{        
    long long int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < no_nodes)
    {
        if(vs[id].inQ == true && vs[id].visited == false)
        {
            vs[id].visited = true;

            vs[id].inQ = false;

            __syncthreads();
            
            Edge temp = es[id];

            for(int i = 0; i < temp.no_to; i++)
            {
                int n_id = temp.tos[i];
                if(vs[n_id].visited == false)
                {
                    *done = false;
                    
                    if(vs[n_id].distance == vs[id].distance + 1)
                    {
                        vs[n_id].parents[vs[n_id].no_parents] = id;
                        vs[n_id].no_parents += 1;
                        vs[n_id].inQ = true;
                    }
                    else if(vs[n_id].distance > vs[id].distance + 1)
                    {
                        vs[n_id].parents[0] = id;
                        vs[n_id].no_parents = 1;
                        vs[n_id].distance = vs[id].distance + 1;
                        vs[n_id].inQ = true;
                    }
                    // printf("Visited and Done %d  %d\n", vs[n_id].visited, done);
                }
            }

            // printf("\n\n");
        }
    }
}
